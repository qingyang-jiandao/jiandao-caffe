#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/util/gpu_util.cuh"
#include "caffe/st_pt_layer.hpp"
#include "caffe/util/benchmark.hpp"

namespace caffe {

template <typename Dtype>
__global__ void set_value_to_constant(const int nthreads, Dtype value, int size, 
	int i, Dtype* dst) {

	CUDA_KERNEL_LOOP(index, nthreads) {
		dst[index * size + i] = value;
	}
}

template <typename Dtype>
__global__ void copy_values(const int nthreads, int size_src, int k, 
	const Dtype* src, int size_dst, int i, Dtype* dst) {

	CUDA_KERNEL_LOOP(index, nthreads) {
		dst[index * size_dst + i] = src[index * size_src + k];
	}
}

template <typename Dtype>
__global__ void SpatialTransformerPTForwardGPU(const int nthreads, int N, int C,
	int output_H_, int output_W_, int H, int W,
	const Dtype* input_grid_data, const Dtype* U, Dtype* V) {
	
	CUDA_KERNEL_LOOP(index, nthreads) {

		const int t = index % output_W_;
		const int s = (index / output_W_) % output_H_;
		const int j = (index / (output_W_ * output_H_)) % C;
		const int i = index / (output_W_ * output_H_ * C);

		const Dtype* coordinates = input_grid_data + (output_H_ * output_W_ * 3) * i;
		const int row_idx = output_W_ * s + t;

		const Dtype pw = coordinates[row_idx * 3 + 2];
	  	const Dtype px = coordinates[row_idx * 3] / pw;
	  	const Dtype py = coordinates[row_idx * 3 + 1] / pw;

	  	const int V_offset = index;

	  	V[V_offset] = (Dtype)0.;

	  	const Dtype x = (px + 1) / 2 * H;
	  	const Dtype y = (py + 1) / 2 * W;

	  	int m, n; Dtype w;
	  	const Dtype* pic = U + i * (C * H * W) + j * (H * W);

	  	m = floor(x); n = floor(y); w = 0;
	  	if(m >= 0 && m < H && n >= 0 && n < W) {
	  		w = (1 - (x - m)) * (1 - (y - n));
	  		V[V_offset] += w * pic[m * W + n];
	  	}

	  	m = floor(x) + 1; n = floor(y); w = 0;
	  	if(m >= 0 && m < H && n >= 0 && n < W) {
	  		w = (1 - (m - x)) * (1 - (y - n));
	  		V[V_offset] += w * pic[m * W + n];
	  	}

	  	m = floor(x); n = floor(y) + 1; w = 0;
	  	if(m >= 0 && m < H && n >= 0 && n < W) {
	  		w = (1 - (x - m)) * (1 - (n - y));
	  		V[V_offset] += w * pic[m * W + n];
	  	}

	  	m = floor(x) + 1; n = floor(y) + 1; w = 0;
	  	if(m >= 0 && m < H && n >= 0 && n < W) {
	  		w = (1 - (m - x)) * (1 - (n - y));
	  		V[V_offset] += w * pic[m * W + n];
	  	} 

		//m = floor(x); n = floor(y); w = (Dtype)0;
		//if (m >= 0 && m < H && n >= 0 && n < W) {
		//	w = max((Dtype)0, (Dtype)1 - abs(x - m)) * max((Dtype)0, (Dtype)1 - abs(y - n));
		//	V[V_offset] += w * pic[m * W + n];
		//}

		//m = floor(x) + 1; n = floor(y); w = (Dtype)0;
		//if (m >= 0 && m < H && n >= 0 && n < W) {
		//	w = max((Dtype)0, (Dtype)1 - abs(x - m)) * max((Dtype)0, (Dtype)1 - abs(y - n));
		//	V[V_offset] += w * pic[m * W + n];
		//}

		//m = floor(x); n = floor(y) + 1; w = (Dtype)0;
		//if (m >= 0 && m < H && n >= 0 && n < W) {
		//	w = max((Dtype)0, (Dtype)1 - abs(x - m)) * max((Dtype)0, (Dtype)1 - abs(y - n));
		//	V[V_offset] += w * pic[m * W + n];
		//}

		//m = floor(x) + 1; n = floor(y) + 1; w = (Dtype)0;
		//if (m >= 0 && m < H && n >= 0 && n < W) {
		//	w = max((Dtype)0, (Dtype)1 - abs(x - m)) * max((Dtype)0, (Dtype)1 - abs(y - n));
		//	V[V_offset] += w * pic[m * W + n];
		//}
  }
}

template <typename Dtype>
__global__ void overflow_test(const int nthreads, int N,
	int output_H_, int output_W_, Dtype* input_grid_data) {

	CUDA_KERNEL_LOOP(index, nthreads) {

		const int t = index % output_W_;
		const int s = (index / output_W_) % output_H_;
		const int i = index / (output_W_ * output_H_);

		//Dtype pw = input_grid_data[3 * index + 2];
		//if (pw < 0.000001 && pw > -0.000001) {
		//	if (pw > 0) {
		//		input_grid_data[3 * index + 2] = 0.0001;
		//	}
		//	else {
		//		input_grid_data[3 * index + 2] = -0.0001;
		//	}
		//}

		Dtype pw = input_grid_data[index * 3 + 2];
		input_grid_data[index * 3] = input_grid_data[index * 3] / pw;
		input_grid_data[index * 3 + 1] = input_grid_data[index * 3 + 1] / pw;
	}
}

template <typename Dtype>
void SpatialTransformerPTLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {

	string prefix = "SpatialTransformerPTLayer::Forward_gpu::\t";
	//Forward_cpu(bottom, top);

	const Dtype* U = bottom[0]->gpu_data();
	const Dtype* theta = bottom[1]->gpu_data();
	Dtype* output_grid_data = output_grid.mutable_gpu_data();

	//std::cout << "output_grid data sync end " << std::endl;
	
	Dtype* full_theta_data = full_theta.mutable_gpu_data();
	Dtype* input_grid_data = input_grid.mutable_gpu_data();
	Dtype* V = top[0]->mutable_gpu_data();

	caffe_gpu_set(input_grid.count(), (Dtype)0, input_grid_data);
	caffe_gpu_set(top[0]->count(), (Dtype)0, V);
	
	// compute full_theta
	int k = 0; 
	const int num_threads = N;
	for(int i=0; i<9; ++i) {
		if (is_pre_defined_theta[i]) {
			set_value_to_constant<Dtype> << <CAFFE_GET_BLOCKS(num_threads), CAFFE_CUDA_NUM_THREADS >> >(
				num_threads, pre_defined_theta[i], 9, i, full_theta_data);
			//std::cout << "Setting value " << pre_defined_theta[i] << " to "<< i << 
			//	"/9 of full_theta_data" << std::endl;
		}
		else {
			copy_values<Dtype> << <CAFFE_GET_BLOCKS(num_threads), CAFFE_CUDA_NUM_THREADS >> >(num_threads,
				9 - pre_defined_count, k, theta, 9, i, full_theta_data);
			//std::cout << "Copying " << k << "/" << 9 - pre_defined_count << " of theta to " 
			//	<< i << "/9 of full_theta_data" << std::endl;
			++k;
		}
	}

	// compute out input_grid_data
	for(int i = 0; i < N; ++i) {
		caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasTrans, output_H_ * output_W_, 3, 3, (Dtype)1.,
				output_grid_data, full_theta_data + 9 * i, (Dtype)0.,
				input_grid_data + (output_H_ * output_W_ * 3) * i);
	}

	test_defined_count = test_defined_count + 1;
	////if (test_defined_count == 1000) 
	//{
	//	std::cout << "dw dw dw" << std::endl;
	//	const Dtype* full_theta_test = full_theta.cpu_data();
	//	for (int index = 0; index < full_theta.count(); ++index) {
	//		Dtype theta = full_theta_test[index];
	//		std::cout << theta << " ";
	//	}
	//	std::cout << std::endl << std::endl;
	//}

	//******be care overfitting.******** no bug
	//const int gpu_nthreads = N * output_H_ * output_W_;
	//overflow_test<Dtype> << <CAFFE_GET_BLOCKS(gpu_nthreads),
	//	CAFFE_CUDA_NUM_THREADS >> >(gpu_nthreads, N, output_H_, output_W_, input_grid_data);

	//if (test_defined_count == 1)
	//{
	//	std::cout << "dw dw dw" << std::endl;
	//	const Dtype* input_grid_data_test = input_grid.cpu_data();
	//	for (int index = 0; index < output_H_ * output_W_; ++index) {
	//		Dtype pw = input_grid_data_test[3 * index + 2];
	//		std::cout << pw << " ";
	//	}
	//	std::cout << std::endl << std::endl;

	//	std::cout << "dx dx dx" << std::endl;
	//	for (int index = 0; index < output_H_ * output_W_; ++index) {
	//		Dtype pw = input_grid_data_test[3 * index];
	//		std::cout << pw << " ";
	//	}
	//	std::cout << std::endl << std::endl;

	//	std::cout << "dy dy dy" << std::endl;
	//	for (int index = 0; index < output_H_ * output_W_; ++index) {
	//		Dtype pw = input_grid_data_test[3 * index + 1];
	//		std::cout << pw << " ";
	//	}
	//	std::cout << std::endl << std::endl;

	//	//Dtype* break_ptr = 0;
	//	//*break_ptr = 1;
	//}

	//std::cout << output_H_ << " " << output_W_ << " " << H << " " << W << " " << N << " " << C << " ";
#if 1
	const int nthreads = N * C * output_H_ * output_W_;
	SpatialTransformerPTForwardGPU<Dtype><<<CAFFE_GET_BLOCKS(nthreads),
	      CAFFE_CUDA_NUM_THREADS>>>(nthreads, N, C, output_H_, output_W_, H, W, input_grid_data, U, V);
#endif

#if 0
	const Dtype* input_grid_cpu_data = input_grid.cpu_data();
	const Dtype* cpu_U = bottom[0]->cpu_data();
	Dtype* cpu_V = top[0]->mutable_cpu_data();

	for (int i = 0; i < N; ++i) {

		const Dtype* coordinates = input_grid_cpu_data + (output_H_ * output_W_ * 3) * i;

		int row_idx; Dtype px, py;

		for (int j = 0; j < C; ++j)
			for (int s = 0; s < output_H_; ++s)
				for (int t = 0; t < output_W_; ++t) {

					row_idx = output_W_ * s + t;

					px = coordinates[row_idx * 3] / coordinates[row_idx * 3 + 2];
					py = coordinates[row_idx * 3 + 1] / coordinates[row_idx * 3 + 2];

					cpu_V[top[0]->offset(i, j, s, t)] = transform_forward_cpu(
						cpu_U + bottom[0]->offset(i, j, 0, 0), px, py);
				}
	}
#endif
}

template <typename Dtype>
__global__ void SpatialTransformerPTBackwardGPU_dTheta(const int nthreads, int C,
		int output_H_, int output_W_, int H, int W,
		const Dtype* input_grid_data, const Dtype* dV_array, const Dtype* U_array,  
		Dtype* dTheta_tmp_diff) {
	
	CUDA_KERNEL_LOOP(index, nthreads) {

		const int t = index % output_W_;
		const int s = (index / output_W_) % output_H_;
		const int j = (index / (output_W_ * output_H_)) % C;
		const int i = index / (output_W_ * output_H_ * C);

		const Dtype* coordinates = input_grid_data + (output_H_ * output_W_ * 3) * i;

		const int row_idx = output_W_ * s + t;

		const Dtype pw = coordinates[row_idx * 3 + 2];
		const Dtype px = coordinates[row_idx * 3] / pw;
		const Dtype py = coordinates[row_idx * 3 + 1] / pw;
		
		Dtype delta_dpx = (Dtype)0.;
		Dtype delta_dpy = (Dtype)0.;
		Dtype delta_dpw = (Dtype)0.;

		const Dtype x = (px + 1) / 2 * H;
		const Dtype y = (py + 1) / 2 * W;
		const int dV_offset = index;
		const Dtype dV = dV_array[dV_offset];

		int m, n; 
		const Dtype* U = U_array + i * (C * H * W) + j * (H * W);

		// left-bottom neighbor
		m = floor(x); n = floor(y); 
		if(m >= 0 && m < H && n >= 0 && n < W) {
			delta_dpx -= (1 - (y - n)) * U[m * W + n] * dV * H / 2;
			delta_dpy -= (1 - (x - m)) * U[m * W + n] * dV * W / 2;
		}
		
		// left-top neighbor
		m = floor(x); n = floor(y) + 1; 
		if(m >= 0 && m < H && n >= 0 && n < W) {
			delta_dpx -= (1 - (n - y)) * U[m * W + n] * dV * H / 2;
			delta_dpy += (1 - (x - m)) * U[m * W + n] * dV * W / 2;
		}

		// right-bottom neighbor
		m = floor(x) + 1; n = floor(y); 
		if(m >= 0 && m < H && n >= 0 && n < W) {
			delta_dpx += (1 - (y - n)) * U[m * W + n] * dV * H / 2;
			delta_dpy -= (1 - (m - x)) * U[m * W + n] * dV * W / 2;
		}
		
		// right-top neighbor
		m = floor(x) + 1; n = floor(y) + 1; 
		if(m >= 0 && m < H && n >= 0 && n < W) {
			delta_dpx += (1 - (n - y)) * U[m * W + n] * dV * H / 2;
			delta_dpy += (1 - (m - x)) * U[m * W + n] * dV * W / 2;
		}
		
		delta_dpw = delta_dpx*(-coordinates[row_idx * 3])/ (pw*pw) + delta_dpy*(-coordinates[row_idx * 3+1]) / (pw*pw);
		//******be care overfitting.********
		//delta_dpw = delta_dpx*(-px)/pw + delta_dpy*(-py) / pw;
		
		int idx = j * (output_H_ * output_W_) + s * output_W_ + t;
		
		dTheta_tmp_diff[(9 * i) * (output_H_ * output_W_ * C) + idx] += delta_dpx * (s * 1.0 / output_H_ * 2 - 1) / pw;
		dTheta_tmp_diff[(9 * i + 1) * (output_H_ * output_W_ * C) + idx] += delta_dpx * (t * 1.0 / output_W_ * 2 - 1) / pw;
		dTheta_tmp_diff[(9 * i + 2) * (output_H_ * output_W_ * C) + idx] += delta_dpx / pw;
		dTheta_tmp_diff[(9 * i + 3) * (output_H_ * output_W_ * C) + idx] += delta_dpy * (s * 1.0 / output_H_ * 2 - 1) / pw;
		dTheta_tmp_diff[(9 * i + 4) * (output_H_ * output_W_ * C) + idx] += delta_dpy * (t * 1.0 / output_W_ * 2 - 1) / pw;
		dTheta_tmp_diff[(9 * i + 5) * (output_H_ * output_W_ * C) + idx] += delta_dpy / pw;
		dTheta_tmp_diff[(9 * i + 6) * (output_H_ * output_W_ * C) + idx] += delta_dpw * (s * 1.0 / output_H_ * 2 - 1);
		dTheta_tmp_diff[(9 * i + 7) * (output_H_ * output_W_ * C) + idx] += delta_dpw * (t * 1.0 / output_W_ * 2 - 1);
		dTheta_tmp_diff[(9 * i + 8) * (output_H_ * output_W_ * C) + idx] += delta_dpw;
	}
}

template <typename Dtype>
__global__ void SpatialTransformerPTBackwardGPU_dU(const int nthreads, const int C, 
	const int W,  const int H, const int output_H_, const int output_W_, 
	const Dtype* input_grid_data, const Dtype* dV, Dtype* dU) {
	
	CUDA_KERNEL_LOOP(index, nthreads) {

		const int t = index % output_W_;
		const int s = (index / output_W_) % output_H_;
		const int j = (index / (output_W_ * output_H_)) % C;
		const int i = index / (output_W_ * output_H_ * C);

		const Dtype* coordinates = input_grid_data + (output_H_ * output_W_ * 3) * i;
		const int row_idx = output_W_ * s + t;

		const Dtype pw = coordinates[row_idx * 3 + 2];
		const Dtype px = coordinates[row_idx * 3] / pw;
		const Dtype py = coordinates[row_idx * 3 + 1] / pw;

	  	const int V_offset = index;

	  	const Dtype x = (px + 1) / 2 * H;
	  	const Dtype y = (py + 1) / 2 * W;

	  	int m, n; Dtype w;
	  	Dtype* pic = dU + i * (C * H * W) + j * (H * W);

	  	m = floor(x); n = floor(y); w = 0;
	  	if(m >= 0 && m < H && n >= 0 && n < W) {
	  		w = (1 - (x - m)) * (1 - (y - n));
			caffe_gpu_atomic_add(w * dV[V_offset], pic + (m * W + n));
	  	}

	  	m = floor(x) + 1; n = floor(y); w = 0;
	  	if(m >= 0 && m < H && n >= 0 && n < W) {
	  		w = (1 - (m - x)) * (1 - (y - n));
			caffe_gpu_atomic_add(w * dV[V_offset], pic + (m * W + n));
	  	}

	  	m = floor(x); n = floor(y) + 1; w = 0;
	  	if(m >= 0 && m < H && n >= 0 && n < W) {
	  		w = (1 - (x - m)) * (1 - (n - y));
			caffe_gpu_atomic_add(w * dV[V_offset], pic + (m * W + n));
	  	}

	  	m = floor(x) + 1; n = floor(y) + 1; w = 0;
	  	if(m >= 0 && m < H && n >= 0 && n < W) {
	  		w = (1 - (m - x)) * (1 - (n - y));
			caffe_gpu_atomic_add(w * dV[V_offset], pic + (m * W + n));
	  	}
	}
}

template <typename Dtype>
void SpatialTransformerPTLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {

	string prefix = "SpatialTransformerPTLayer::Backward_GPU::\t";

	const Dtype* dV = top[0]->gpu_diff();
	const Dtype* input_grid_data = input_grid.gpu_data();
	const Dtype* U = bottom[0]->gpu_data();

	Dtype* dFull_theta = full_theta.mutable_gpu_diff();
	Dtype* dTheta = bottom[1]->mutable_gpu_diff();
	Dtype* dTheta_tmp_diff = dTheta_tmp.mutable_gpu_diff();

	caffe_gpu_set(dTheta_tmp.count(), (Dtype)0., dTheta_tmp_diff);

	const int nthreads = N * C * output_H_ * output_W_;

	SpatialTransformerPTBackwardGPU_dTheta<Dtype><<<CAFFE_GET_BLOCKS(nthreads),
			CAFFE_CUDA_NUM_THREADS>>>(nthreads, C, output_H_, output_W_, H, W, input_grid_data,
					dV, U, dTheta_tmp_diff);

	Dtype* all_ones_2_data = all_ones_2.mutable_gpu_data();
	caffe_gpu_set(all_ones_2.count(), (Dtype)1., all_ones_2_data);
	
	caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, full_theta.count(), 1, output_H_ * output_W_ * C, 
			(Dtype)1., dTheta_tmp_diff, all_ones_2_data, (Dtype)0., dFull_theta);
			
	/*const Dtype* db_dFull_theta = full_theta.cpu_diff();
	for(int i=0; i<full_theta.count(); ++i) {
		std::cout << db_dFull_theta[i] << " ";
	}
	std::cout<<std::endl;*/
			
	int k = 0;
	const int num_threads = N;
	for(int i=0; i<9; ++i) {
		if (!is_pre_defined_theta[i]) {
			copy_values<Dtype> << <CAFFE_GET_BLOCKS(num_threads), CAFFE_CUDA_NUM_THREADS >> >(num_threads,
				9, i, dFull_theta, 9 - pre_defined_count, k, dTheta);
			//std::cout << "Copying " << i << "/9 of dFull_theta to " << k << "/" << 
			//	9 - pre_defined_count << " of dTheta" << std::endl;
			++k;
		}
	}
	
	/*const Dtype* db_dtheta = bottom[1]->cpu_diff();
	for(int i=0; i<bottom[1]->count(); ++i) {
		std::cout << db_dtheta[i] << " ";
	}
	std::cout<<std::endl;*/
			
	if(to_compute_dU_) {
		Dtype* dU = bottom[0]->mutable_gpu_diff();
		caffe_gpu_set(bottom[0]->count(), (Dtype)0., dU);
		const int nthreads = N * C * output_H_ * output_W_;
		SpatialTransformerPTBackwardGPU_dU<Dtype><<<CAFFE_GET_BLOCKS(nthreads),
			CAFFE_CUDA_NUM_THREADS>>>(nthreads, C, W, H, output_H_, output_W_, input_grid_data, dV, dU);
	}
}

INSTANTIATE_LAYER_GPU_FUNCS(SpatialTransformerPTLayer);

}	// namespace caffe
