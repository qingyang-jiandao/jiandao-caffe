#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/util/gpu_util.cuh"
#include "caffe/st_pt_op_layer.hpp"
#include "caffe/util/benchmark.hpp"

namespace caffe {

template <typename Dtype>
__global__ void set_value_to_constant(const int nthreads, Dtype value, int size, 
	int i, Dtype* dst) {

	CUDA_KERNEL_LOOP(index, nthreads) {
		dst[index * size + i] = value;
	}
}

template <typename Dtype>
__global__ void copy_values(const int nthreads, int size_src, int k, 
	const Dtype* src, int size_dst, int i, Dtype* dst) {

	CUDA_KERNEL_LOOP(index, nthreads) {
		dst[index * size_dst + i] = src[index * size_src + k];
	}
}

template <typename Dtype>
__global__ void SpatialTransformerForwardGPU(const int nthreads, int N, int C,
	int output_H_, int output_W_, int H, int W,
	const Dtype* input_grid_data, const Dtype* all_theta, const Dtype* U, Dtype* V) {

	CUDA_KERNEL_LOOP(index, nthreads) {

		const int t = index % output_W_;
		const int s = (index / output_W_) % output_H_;
		const int j = (index / (output_W_ * output_H_)) % C;
		const int i = index / (output_W_ * output_H_ * C);

		const Dtype centerpoint_offset_x = all_theta[11 * i + 9];
		const Dtype centerpoint_offset_y = all_theta[11 * i + 10];

		const Dtype* coordinates = input_grid_data + (output_H_ * output_W_ * 3) * i;
		const int row_idx = output_W_ * s + t;

		const Dtype pw = coordinates[row_idx * 3 + 2];
		const Dtype px = coordinates[row_idx * 3] / pw;
		const Dtype py = coordinates[row_idx * 3 + 1] / pw;

		const int V_offset = index;

		V[V_offset] = (Dtype)0.;

		const Dtype x = (px + centerpoint_offset_x) / 2 * H;
		const Dtype y = (py + centerpoint_offset_y) / 2 * W;

		int m, n; Dtype w;
		const Dtype* pic = U + i * (C * H * W) + j * (H * W);

		m = floor(x); n = floor(y); w = 0;
		if (m >= 0 && m < H && n >= 0 && n < W) {
			w = (1 - (x - m)) * (1 - (y - n));
			V[V_offset] += w * pic[m * W + n];
		}

		m = floor(x) + 1; n = floor(y); w = 0;
		if (m >= 0 && m < H && n >= 0 && n < W) {
			w = (1 - (m - x)) * (1 - (y - n));
			V[V_offset] += w * pic[m * W + n];
		}

		m = floor(x); n = floor(y) + 1; w = 0;
		if (m >= 0 && m < H && n >= 0 && n < W) {
			w = (1 - (x - m)) * (1 - (n - y));
			V[V_offset] += w * pic[m * W + n];
		}

		m = floor(x) + 1; n = floor(y) + 1; w = 0;
		if (m >= 0 && m < H && n >= 0 && n < W) {
			w = (1 - (m - x)) * (1 - (n - y));
			V[V_offset] += w * pic[m * W + n];
		}
	}
}

template <typename Dtype>
__global__ void overflow_test(const int nthreads, int N,
	int output_H_, int output_W_, Dtype* output_grid_data) {

	CUDA_KERNEL_LOOP(index, nthreads) {

		const int t = index % output_W_;
		const int s = (index / output_W_) % output_H_;
		const int i = index / (output_W_ * output_H_);

		Dtype pw = output_grid_data[3 * index + 2];
		if (pw < 0.000001 && pw > -0.000001) {
			if (pw > 0) {
				output_grid_data[3 * index + 2] = 0.0001;
			}
			else {
				output_grid_data[3 * index + 2] = -0.0001;
			}
		}
	}
}

template <typename Dtype>
__global__ void move_grid(const int nthreads, int N,
	int output_H_, int output_W_, const Dtype* all_theta, Dtype* inner_grid_data) {

	CUDA_KERNEL_LOOP(index, nthreads) {

		const int t = index % output_W_;
		const int s = (index / output_W_) % output_H_;
		const int i = index / (output_W_ * output_H_);

		const Dtype centerpoint_offset_x = all_theta[11 * i + 9];
		const Dtype centerpoint_offset_y = all_theta[11 * i + 10];

		const int row_idx = output_W_ * s + t;
		//Dtype* curr_inner_grid_data = inner_grid_data + (output_H_ * output_W_ * 3) * i;
		//curr_inner_grid_data[row_idx] = s * 1.0 / output_H_ * 2 - centerpoint_offset_x;
		//curr_inner_grid_data[row_idx+1] = t * 1.0 / output_W_ * 2 - centerpoint_offset_y;
		//curr_inner_grid_data[row_idx + 2] = 1;

		inner_grid_data[3 * index] = s * 1.0 / output_H_ * 2 - centerpoint_offset_x;
		inner_grid_data[3 * index + 1] = t * 1.0 / output_W_ * 2 - centerpoint_offset_y;
		inner_grid_data[3 * index + 2] = 1;
	}
}

template <typename Dtype>
void SpatialTransformerPTOPLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {

	string prefix = "SpatialTransformerPTOPLayer::Forward_gpu::\t";

	const Dtype* U = bottom[0]->gpu_data();
	const Dtype* theta = bottom[1]->gpu_data();
	const Dtype* output_grid_data = output_grid.mutable_gpu_data();
	const int thete_count = bottom[1]->shape(1);

	//std::cout << "output_grid data sync end " << std::endl;
	
	Dtype* full_theta_data = full_theta.mutable_gpu_data();
	Dtype* input_grid_data = input_grid.mutable_gpu_data();
	Dtype* inner_grid_data = inner_grid.mutable_gpu_data();
	Dtype* V = top[0]->mutable_gpu_data();

	caffe_gpu_set(input_grid.count(), (Dtype)0, input_grid_data);
	caffe_gpu_set(top[0]->count(), (Dtype)0, V);
	
	// compute full_theta
	int k = 0; 
	const int num_threads = N;
	for(int i=0; i<9; ++i) {
		if (is_pre_defined_theta[i]) {
			set_value_to_constant<Dtype> << <CAFFE_GET_BLOCKS(num_threads), CAFFE_CUDA_NUM_THREADS >> > (
				num_threads, pre_defined_theta[i], 9, i, full_theta_data);
			//std::cout << "Setting value " << pre_defined_theta[i] << " to "<< i << 
			//	"/9 of full_theta_data" << std::endl;
		}
		else {
			copy_values<Dtype> << <CAFFE_GET_BLOCKS(num_threads), CAFFE_CUDA_NUM_THREADS >> > (num_threads,
				11 - pre_defined_count, k, theta, 9, i, full_theta_data);
			//std::cout << "Copying " << k << "/" << 9 - pre_defined_count << " of theta to " 
			//	<< i << "/9 of full_theta_data" << std::endl;
			++k;
		}
	}

	const int gpu_nthreads = N * output_H_ * output_W_;
	move_grid<Dtype> << <CAFFE_GET_BLOCKS(gpu_nthreads),
		CAFFE_CUDA_NUM_THREADS >> >(gpu_nthreads, N, output_H_, output_W_, theta, inner_grid_data);

	// compute out input_grid_data
	for(int i = 0; i < N; ++i) {
		Dtype* curr_inner_grid_data = inner_grid_data + (output_H_ * output_W_ * 3) * i;
		caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasTrans, output_H_ * output_W_, 3, 3, (Dtype)1.,
			curr_inner_grid_data, full_theta_data + 9 * i, (Dtype)0.,
				input_grid_data + (output_H_ * output_W_ * 3) * i);
	}

	//for(int i = 0; i < N; ++i) {
	//	caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasTrans, output_H_ * output_W_, 3, 3, (Dtype)1.,
	//			output_grid_data, full_theta_data + 9 * i, (Dtype)0.,
	//			input_grid_data + (output_H_ * output_W_ * 3) * i);
	//}

	//test_defined_count = test_defined_count + 1;
	//if (test_defined_count == 1000) {
	//	std::cout << "dw dw dw" << std::endl;
	//	const Dtype* input_grid_data_test = input_grid.cpu_data();
	//	for (int index = 0; index < output_H_ * output_W_; ++index) {
	//		Dtype pw = input_grid_data_test[3 * index + 2];
	//		std::cout << pw << " ";
	//	}
	//	std::cout << std::endl << std::endl;
	//}

	//******be care overfitting.********
	//const int gpu_nthreads = N * output_H_ * output_W_;
	//overflow_test<Dtype> << <CAFFE_GET_BLOCKS(gpu_nthreads),
	//	CAFFE_CUDA_NUM_THREADS >> >(gpu_nthreads, N, output_H_, output_W_, input_grid_data);

	//if (test_defined_count == 1000) {
	//	std::cout << "dw dw dw" << std::endl;
	//	const Dtype* input_grid_data_test = input_grid.cpu_data();
	//	for (int index = 0; index < output_H_ * output_W_; ++index) {
	//		Dtype pw = input_grid_data_test[3 * index + 2];
	//		std::cout << pw << " ";
	//	}
	//	std::cout << std::endl << std::endl;

	//	Dtype* break_ptr = 0;
	//	*break_ptr = 1;
	//}

	const int nthreads = N * C * output_H_ * output_W_;

	SpatialTransformerForwardGPU<Dtype><<<CAFFE_GET_BLOCKS(nthreads),
	      CAFFE_CUDA_NUM_THREADS>>>(nthreads, N, C, output_H_, output_W_, H, W, input_grid_data, theta, U, V);
}

template <typename Dtype>
__global__ void SpatialTransformerBackwardGPU_dTheta(const int nthreads, int C,
		int output_H_, int output_W_, int H, int W,
		const Dtype* input_grid_data, const Dtype* all_theta, const Dtype* inner_grid_data, const Dtype* dV_array, const Dtype* U_array,
		Dtype* dTheta_tmp_diff, Dtype* dTheta_tmp_point_xy_diff) {
	
	CUDA_KERNEL_LOOP(index, nthreads) {

		const int t = index % output_W_;
		const int s = (index / output_W_) % output_H_;
		const int j = (index / (output_W_ * output_H_)) % C;
		const int i = index / (output_W_ * output_H_ * C);

		const Dtype* i_offsets = all_theta + 11 * i;
		const Dtype centerpoint_offset_x = i_offsets[9];
		const Dtype centerpoint_offset_y = i_offsets[10];

		const int row_idx = output_W_ * s + t;
		const Dtype* coordinates = input_grid_data + (output_H_ * output_W_ * 3) * i;
		const Dtype pw = coordinates[row_idx * 3 + 2];
		const Dtype px = coordinates[row_idx * 3] / pw;
		const Dtype py = coordinates[row_idx * 3 + 1] / pw;

		const Dtype* inner_grid_coordinates = inner_grid_data + (output_H_ * output_W_ * 3) * i;
		const Dtype inner_px = inner_grid_coordinates[row_idx * 3];
		const Dtype inner_py = inner_grid_coordinates[row_idx * 3 + 1];
		
		Dtype delta_dpx = (Dtype)0.;
		Dtype delta_dpy = (Dtype)0.;
		Dtype delta_dpw = (Dtype)0.;

		const Dtype x = (px + centerpoint_offset_x) / 2 * H;
		const Dtype y = (py + centerpoint_offset_y) / 2 * W;
		const int dV_offset = index;
		const Dtype dV = dV_array[dV_offset];

		int m, n; 
		const Dtype* U = U_array + i * (C * H * W) + j * (H * W);

		m = floor(x); n = floor(y); 
		if(m >= 0 && m < H && n >= 0 && n < W) {
			delta_dpx -= (1 - (y - n)) * U[m * W + n] * dV * H / 2;
			delta_dpy -= (1 - (x - m)) * U[m * W + n] * dV * W / 2;
		}
		
		m = floor(x); n = floor(y) + 1; 
		if(m >= 0 && m < H && n >= 0 && n < W) {
			delta_dpx -= (1 - (n - y)) * U[m * W + n] * dV * H / 2;
			delta_dpy += (1 - (x - m)) * U[m * W + n] * dV * W / 2;
		}

		m = floor(x) + 1; n = floor(y); 
		if(m >= 0 && m < H && n >= 0 && n < W) {
			delta_dpx += (1 - (y - n)) * U[m * W + n] * dV * H / 2;
			delta_dpy -= (1 - (m - x)) * U[m * W + n] * dV * W / 2;
		}
		
		m = floor(x) + 1; n = floor(y) + 1; 
		if(m >= 0 && m < H && n >= 0 && n < W) {
			delta_dpx += (1 - (n - y)) * U[m * W + n] * dV * H / 2;
			delta_dpy += (1 - (m - x)) * U[m * W + n] * dV * W / 2;
		}
		
		delta_dpw = delta_dpx*(-coordinates[row_idx * 3])/ (pw*pw) + delta_dpy*(-coordinates[row_idx * 3+1]) / (pw*pw);
		//******be care overfitting.********
		//delta_dpw = delta_dpx*(-px)/pw + delta_dpy*(-py) / pw;

		int delta_centerpoint_offset_x = delta_dpx + delta_dpx*(-pw*i_offsets[0] / (pw*pw) + coordinates[row_idx * 3] * i_offsets[6] / (pw*pw)) + delta_dpy*(-pw*i_offsets[3] / (pw*pw) + coordinates[row_idx * 3 + 1] * i_offsets[6] / (pw*pw));
		int delta_centerpoint_offset_y = delta_dpy + delta_dpx*(-pw*i_offsets[1] / (pw*pw) + coordinates[row_idx * 3] * i_offsets[7] / (pw*pw)) + delta_dpy*(-pw*i_offsets[4] / (pw*pw) + coordinates[row_idx * 3 + 1] * i_offsets[7] / (pw*pw));

		int idx = j * (output_H_ * output_W_) + s * output_W_ + t;
		
		dTheta_tmp_diff[(9 * i) * (output_H_ * output_W_ * C) + idx] += delta_dpx * inner_px / pw;
		dTheta_tmp_diff[(9 * i + 1) * (output_H_ * output_W_ * C) + idx] += delta_dpx * inner_py / pw;
		dTheta_tmp_diff[(9 * i + 2) * (output_H_ * output_W_ * C) + idx] += delta_dpx / pw;
		dTheta_tmp_diff[(9 * i + 3) * (output_H_ * output_W_ * C) + idx] += delta_dpy * inner_px / pw;
		dTheta_tmp_diff[(9 * i + 4) * (output_H_ * output_W_ * C) + idx] += delta_dpy * inner_py / pw;
		dTheta_tmp_diff[(9 * i + 5) * (output_H_ * output_W_ * C) + idx] += delta_dpy / pw;
		dTheta_tmp_diff[(9 * i + 6) * (output_H_ * output_W_ * C) + idx] += delta_dpw * inner_px;
		dTheta_tmp_diff[(9 * i + 7) * (output_H_ * output_W_ * C) + idx] += delta_dpw * inner_py;
		dTheta_tmp_diff[(9 * i + 8) * (output_H_ * output_W_ * C) + idx] += delta_dpw;

		dTheta_tmp_point_xy_diff[(2 * i) * (output_H_ * output_W_ * C) + idx] += delta_centerpoint_offset_x;
		dTheta_tmp_point_xy_diff[(2 * i + 1) * (output_H_ * output_W_ * C) + idx] += delta_centerpoint_offset_y;
	}
}

template <typename Dtype>
__global__ void SpatialTransformerBackwardGPU_dU(const int nthreads, const int C, 
	const int W,  const int H, const int output_H_, const int output_W_, 
	const Dtype* input_grid_data, const Dtype* all_theta, const Dtype* dV, Dtype* dU) {
	
	CUDA_KERNEL_LOOP(index, nthreads) {

		const int t = index % output_W_;
		const int s = (index / output_W_) % output_H_;
		const int j = (index / (output_W_ * output_H_)) % C;
		const int i = index / (output_W_ * output_H_ * C);

		const Dtype* i_offsets = all_theta + 11 * i;
		const Dtype centerpoint_offset_x = i_offsets[9];
		const Dtype centerpoint_offset_y = i_offsets[10];

		const Dtype* coordinates = input_grid_data + (output_H_ * output_W_ * 3) * i;
		const int row_idx = output_W_ * s + t;

		const Dtype pw = coordinates[row_idx * 3 + 2];
		const Dtype px = coordinates[row_idx * 3] / pw;
		const Dtype py = coordinates[row_idx * 3 + 1] / pw;

	  	const int V_offset = index;

	  	const Dtype x = (px + centerpoint_offset_x) / 2 * H;
	  	const Dtype y = (py + centerpoint_offset_y) / 2 * W;

	  	int m, n; Dtype w;
	  	Dtype* pic = dU + i * (C * H * W) + j * (H * W);

	  	m = floor(x); n = floor(y); w = 0;
	  	if(m >= 0 && m < H && n >= 0 && n < W) {
	  		w = (1 - (x - m)) * (1 - (y - n));
			caffe_gpu_atomic_add(w * dV[V_offset], pic + (m * W + n));
	  	}

	  	m = floor(x) + 1; n = floor(y); w = 0;
	  	if(m >= 0 && m < H && n >= 0 && n < W) {
	  		w = (1 - (m - x)) * (1 - (y - n));
			caffe_gpu_atomic_add(w * dV[V_offset], pic + (m * W + n));
	  	}

	  	m = floor(x); n = floor(y) + 1; w = 0;
	  	if(m >= 0 && m < H && n >= 0 && n < W) {
	  		w = (1 - (x - m)) * (1 - (n - y));
			caffe_gpu_atomic_add(w * dV[V_offset], pic + (m * W + n));
	  	}

	  	m = floor(x) + 1; n = floor(y) + 1; w = 0;
	  	if(m >= 0 && m < H && n >= 0 && n < W) {
	  		w = (1 - (m - x)) * (1 - (n - y));
			caffe_gpu_atomic_add(w * dV[V_offset], pic + (m * W + n));
	  	}
	}
}

template <typename Dtype>
__global__ void centerpoint_offset_loss(const int nthreads, int N,
	int output_H_, int output_W_, const Dtype* theta, Dtype* loss_array) {

	CUDA_KERNEL_LOOP(index, nthreads) {

		const Dtype centerpoint_offset_x = theta[11 * index + 9];
		const Dtype centerpoint_offset_y = theta[11 * index + 10];

		Dtype d_x = (Dtype)0, d_y = (Dtype)0;

		if (centerpoint_offset_x < 0) {
			d_x = (centerpoint_offset_x);
		}
		else if (centerpoint_offset_x > 2) {
			d_x = (centerpoint_offset_x - 2);
		}

		if (centerpoint_offset_y < 0) {
			d_y = (centerpoint_offset_y);
		}
		else if (centerpoint_offset_y > 2) {
			d_y = (centerpoint_offset_y - 2);
		}

		loss_array[11 * index + 9] = d_x;
		loss_array[11 * index + 10] = d_y;
	}
}

template <typename Dtype>
void SpatialTransformerPTOPLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {

	string prefix = "SpatialTransformerPTOPLayer::Backward_GPU::\t";

	const Dtype* dV = top[0]->gpu_diff();
	const Dtype* input_grid_data = input_grid.gpu_data();
	const Dtype* inner_grid_data = inner_grid.gpu_data();
	const Dtype* U = bottom[0]->gpu_data();
	const Dtype* theta = bottom[1]->gpu_data();
	const int thete_count = bottom[1]->shape(1);

	Dtype* dFull_theta_diff = full_theta.mutable_gpu_diff();
	Dtype* dFull_theta_point_xy_diff = full_theta_point_xy.mutable_gpu_diff();
	Dtype* dTheta = bottom[1]->mutable_gpu_diff();
	Dtype* dTheta_tmp_diff = dTheta_tmp.mutable_gpu_diff();
	Dtype* dTheta_tmp_point_xy_diff = dTheta_tmp_point_xy.mutable_gpu_diff();

	caffe_gpu_set(dTheta_tmp.count(), (Dtype)0., dTheta_tmp_diff);
	caffe_gpu_set(dTheta_tmp_point_xy.count(), (Dtype)0., dTheta_tmp_point_xy_diff);

	const int nthreads = N * C * output_H_ * output_W_;
	SpatialTransformerBackwardGPU_dTheta<Dtype><<<CAFFE_GET_BLOCKS(nthreads),
			CAFFE_CUDA_NUM_THREADS>>>(nthreads, C, output_H_, output_W_, H, W, input_grid_data, theta, inner_grid_data,
					dV, U, dTheta_tmp_diff, dTheta_tmp_point_xy_diff);

	Dtype* all_ones_2_data = all_ones_2.mutable_gpu_data();
	caffe_gpu_set(all_ones_2.count(), (Dtype)1., all_ones_2_data);
	
	caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, full_theta.count(), 1, output_H_ * output_W_ * C,
			(Dtype)1., dTheta_tmp_diff, all_ones_2_data, (Dtype)0., dFull_theta_diff);

	caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, full_theta_point_xy.count(), 1, output_H_ * output_W_ * C,
		(Dtype)1., dTheta_tmp_point_xy_diff, all_ones_2_data, (Dtype)0., dFull_theta_point_xy_diff);
			
	/*const Dtype* db_dFull_theta = full_theta.cpu_diff();
	for(int i=0; i<full_theta.count(); ++i) {
		std::cout << db_dFull_theta[i] << " ";
	}
	std::cout<<std::endl;*/
			
	int k = 0;
	const int num_threads = N;
	for (int i = 0; i < 11; ++i) {
		if (!is_pre_defined_theta[i] && i<9) {
			copy_values<Dtype> << <CAFFE_GET_BLOCKS(num_threads), CAFFE_CUDA_NUM_THREADS >> > (num_threads,
				9, i, dFull_theta_diff, 11 - pre_defined_count, k, dTheta);
			//std::cout << "Copying " << i << "/9 of dFull_theta to " << k << "/" << 
			//	9 - pre_defined_count << " of dTheta" << std::endl;
			++k;
		}
		else {
			if (!is_pre_defined_theta[i]) {
				copy_values<Dtype> << <CAFFE_GET_BLOCKS(num_threads), CAFFE_CUDA_NUM_THREADS >> > (num_threads,
					2, i-9, dFull_theta_point_xy_diff, 11 - pre_defined_count, k, dTheta);
				//std::cout << "Copying " << i << "/9 of dFull_theta to " << k << "/" << 
				//	9 - pre_defined_count << " of dTheta" << std::endl;
				++k;
			}
		}
	}
	
	//centerpoint_offset_loss<Dtype> << <CAFFE_GET_BLOCKS(num_threads),
	//	CAFFE_CUDA_NUM_THREADS >> >(num_threads, N, output_H_, output_W_, theta, dTheta);
	
	/*const Dtype* db_dtheta = bottom[1]->cpu_diff();
	for(int i=0; i<bottom[1]->count(); ++i) {
		std::cout << db_dtheta[i] << " ";
	}
	std::cout<<std::endl;*/
			
	if(to_compute_dU_) {
		Dtype* dU = bottom[0]->mutable_gpu_diff();
		caffe_gpu_set(bottom[0]->count(), (Dtype)0., dU);
		const int nthreads = N * C * output_H_ * output_W_;
		SpatialTransformerBackwardGPU_dU<Dtype><<<CAFFE_GET_BLOCKS(nthreads),
			CAFFE_CUDA_NUM_THREADS>>>(nthreads, C, W, H, output_H_, output_W_, input_grid_data, theta, dV, dU);
	}
}

INSTANTIATE_LAYER_GPU_FUNCS(SpatialTransformerPTOPLayer);

}	// namespace caffe
